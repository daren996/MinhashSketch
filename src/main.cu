#include "hip/hip_runtime.h"
#include <algorithm>
#include <fstream>
#include <iostream>
#include <iomanip>
#include <list>
#include <random>
#include <vector>
#include <string>

#include "MinhashSketch.cu"
#include "Hash.h"
#include "Utils.h"

using namespace std;

void output_signature(vector <vector<uint64>> sig1) {
    for (int h = 0; h < sig1.size(); ++h) {
        cout << "sig[" << h << "].size(): " << sig1[h].size() << "\t";
        for (int i = 0; i < sig1[h].size(); ++i) {
            cout << hex << sig1[h][i] << dec << " ";
        }
        cout << endl;
    }
}

void usage() {
    cout << "===========================" << endl;
    cerr << "Usage: " << endl << endl;
    cerr << "    ./MinhashSketch FILE_ONE FILE_TWO MODE" << endl;
    cerr << endl;
    cerr << "    Possible MODEs are:" << endl;
    cerr << endl << bold_on;
    cerr << "        all" << endl;
    cerr << endl;
    cerr << "        minhash_parallel" << endl;
    cerr << endl;
    cerr << "Execute \"MinhashSketch help\" for an extended help section." << endl;
    cout << "===========================" << endl;
    exit(1);
}

void help() {
    cout << endl;
    cout << bold_on << "NAME" << bold_off << endl;
    cout << "    " << "MinhashSketch" << endl;
    cout << endl;
    cout << bold_on << "USAGE" << bold_off << endl;
    cout << "    " << "MinhashSketch FILE_ONE FILE_TWO " << bold_on << "MODE [PARAMETERS...]" << bold_off << endl;
    cout << endl;
    cout << "    " << "MinhashSketch calculates the similarity between two text files FILE_ONE and FILE_TWO" << endl;
    cout << "    " << "and outputs it as a number between 0 and 1, where 1 means the two files are exactly" << endl;
    cout << "    " << "the same." << endl;
    cout << endl;
    cout << bold_on << "MODE" << bold_off << endl;
    cout << "    " << "There are modes which change the way MinhashSketch computes the similarity. " << endl;
    cout << "    " << "Each may make use of different parameters, indicated as follows:" << endl;;
    cout << endl;
    cout << "    " << bold_on << "all" << bold_off << endl;
    cout << "        " << "This option executes all modes." << endl;
    cout << endl;
    cout << "    " << bold_on << "minhash" << bold_off << endl;
    cout << "        " << "Calculates the similarity by computing minhash signatures for each sequence. Used" << endl;
    cout << "        " << "parameters are." << endl;
    cout << endl;
    cout << "            " << "--k=POSITIVE_INTEGER as shingle size" << endl;
    cout << endl;
    cout << "            " << "--t=POSITIVE_INTEGER" << bold_on << " (obligatory) " << bold_off
         << "as number of hash functions used" << endl;
    cout << endl;
    cout << "            " << "--seed=INTEGER as random generator seed" << endl;
    cout << endl;
    cout << bold_on << "PARAMETERS" << bold_off << endl;
    cout << endl;
    cout << "    " << bold_on << "--k=POSITIVE_INTEGER" << bold_off << endl;
    cout << "        " << "Defaults to k=9. Indicates the size of the shingles used to calculate the simi-" << endl;
    cout << "        " << "larity between the documents." << endl;
    cout << endl;
    cout << "    " << bold_on << "--m=POSITIVE_INTEGER" << bold_off << endl;
    cout << "        " << "Defaults to m=1. Indicates the number of sketches saved in minhash modes." << endl;
    cout << endl;
    cout << "    " << bold_on << "--t=POSITIVE_INTEGER" << bold_off << endl;
    cout << "        " << "Defaults to t=1. Indicates the number of hash functions used in minhash modes." << endl;
    cout << endl;
    cout << "    " << bold_on << "--seed=INTEGER" << bold_off << endl;
    cout << "        " << "Defaults to a random value. Used by minhash modes in their random generator number." << endl;
    cout << endl;
    cout << "    " << bold_on << "-e" << bold_off << endl;
    cout << "        " << "Output in experimentation format." << endl;
    cout << endl;
    exit(0);
}

/*
void getList(int k, uint64 *list[], string &sequence, vector <Hash> &hashes) {
    for (int j = 0; j < hashes.size(); j++) {
        uint64 begin = 0, end = sequence.size() - k + 1;
        uint64 length = end - begin + 1;
        uint64 s_index = 0; // pointer of current base
        uint64 l_index = 0; // pointer of current list
        uint64 cur_seq[k / 32 + 1]; // current sub-sequence
        for (int i = 0; i < k / 32 + 1; ++i) {
            cur_seq[i] = 0;
        }

        // Get original list
        if (k < 32) {
            for (; s_index < k; ++s_index) {
                if (utils::base2int(sequence[s_index + begin]) != -1)
                    cur_seq[0] =
                            (cur_seq[0] << 2) % ((uint64) 1 << (2 * k)) + utils::base2int(sequence[s_index + begin]);
                else
                    cerr << "ERROR:" << endl << "\t index: " << s_index + begin << endl << "\t base: "
                         << sequence[s_index + begin] << endl;
            }
            list[j][l_index++] = hashes[j](cur_seq);
            for (; s_index < length; ++s_index) {
                if (utils::base2int(sequence[s_index + begin]) != -1)
                    cur_seq[0] =
                            (cur_seq[0] << 2) % ((uint64) 1 << (2 * k)) + utils::base2int(sequence[s_index + begin]);
                else
                    cerr << "ERROR:" << endl << "\t index: " << s_index + begin << endl << "\t base: "
                         << sequence[s_index + begin] << endl;
                list[j][l_index++] = hashes[j](cur_seq);
            }
        } else {
            for (; s_index < k; ++s_index) {
                if (utils::base2int(sequence[s_index + begin]) != -1)
                    cur_seq[s_index / 32] =
                            (cur_seq[s_index / 32] << 2) % UINT64_MAX + utils::base2int(sequence[s_index + begin]);
                else
                    cerr << "ERROR:" << endl << "\t index: " << s_index + begin << endl << "\t base: "
                         << sequence[s_index + begin] << endl;
            }
            list[j][l_index++] = hashes[j](cur_seq);
            for (; s_index < length; ++s_index) {
                for (int i = 0; i < k / 32 - 1; ++i) {
                    cur_seq[i] = (cur_seq[i] << 2) + (cur_seq[i + 1] >> 62);
                }
                cur_seq[k / 32 - 1] = (cur_seq[k / 32 - 1] << 2) + (cur_seq[k / 32] >> ((k % 32) * 2 - 2));
                if (utils::base2int(sequence[s_index + begin]) != -1)
                    cur_seq[k / 32] = (cur_seq[k / 32] << 2) % ((uint64) 1 << (2 * (k % 32))) +
                                      utils::base2int(sequence[s_index + begin]);
                else
                    cerr << "ERROR:" << endl << "\t index: " << s_index + begin << endl << "\t base: "
                         << sequence[s_index + begin] << endl;
                list[j][l_index] = hashes[j](cur_seq);
            }
        }
    }
}
*/

// MinhashSketch.exe ../testing_files/sequence_clip1.fasta ../testing_files/sequence_clip2.fasta all -e --k=5 --m=10 --t=10
int main(int argc, char *argv[]) {

    if (argc == 2 && string(argv[1]) == "help") help();
    if (argc < 4) usage();

    // DEFAULT VALUES
    int k, m, t, seed;
    bool e;
    k = 9;
    m = 1; // the number of sketches
    t = 1; // the number of hash functions
    seed = random_device()();
    e = false;

    // PARSE FILE_ONE FILE_TWO MODE
    string name_one = string(argv[1]);
    string name_two = string(argv[2]);
    string cal_name = string(argv[3]);
    ifstream file1(name_one);
    if (file1.fail()) {
        std::cerr << "Unable to open file " << name_one << std::endl;
        exit(1);
    }
    ifstream file2(name_two);
    if (file2.fail()) {
        std::cerr << "Unable to open file " << name_two << std::endl;
        exit(1);
    }

    // PARSE PARAMETERS
    for (int i = 4; i < argc; ++i) {
        string param(argv[i]);
        if (param == "-e") {
            e = true;
        } else {
            int param_size = (uint) param.size();
            if (param_size >= 5) {
                auto index_eq = (uint) param.find('=');
                if (index_eq + 2 <= param_size) {
                    string param_name = param.substr(0, index_eq);
                    if (param_name == "--k") {
                        k = std::stoi(param.substr(index_eq + 1, param_size - index_eq - 1));
                    } else if (param_name == "--m") {
                        m = std::stoi(param.substr(index_eq + 1, param_size - index_eq - 1));
                    } else if (param_name == "--t") {
                        t = std::stoi(param.substr(index_eq + 1, param_size - index_eq - 1));
                    } else if (param_name == "--seed") {
                        seed = std::stoi(param.substr(index_eq + 1, param_size - index_eq - 1));
                    }
                }
            }
        }
    }
    if (k < 1) {
        std::cerr << "K value too small! Minimum: 1" << std::endl;
        exit(1);
    }
    if (m < 1) {
        std::cerr << "M value too small! Minimum: 1" << std::endl;
        exit(1);
    }

    // GET TWO SEQUENCES
    string file_info1, file_info2, sequence1, sequence2, s1, s2;
    utils::file_to_string(file1, file_info1, sequence1); // The first line is file information
    utils::file_to_string(file2, file_info2, sequence2);
    uint64 sequence_size1 = sequence1.size(), sequence_size2 = sequence2.size();
    if (sequence1.size() < k || sequence2.size() < k) {
        cout << "k cannot be greater than the size of any document" << endl;
        exit(1);
    }
    file1.close();
    file2.close();
    cout << "sequence1.size()" << sequence1.size() << endl;
    cout << "sequence2.size()" << sequence2.size() << endl;
    char dnaList1[sequence1.size()];
    char dnaList2[sequence2.size()];
//    for (int i = 0; i < sequence1.size(); i++)
//        dnaList1[i] = sequence1[i];
//    for (int i = 0; i < sequence2.size(); i++)
//        dnaList2[i] = sequence2[i];
    strcpy(dnaList1, sequence1.c_str());
    strcpy(dnaList2, sequence1.c_str());

    // MAIN PROGRESS
    clock_t ini_time;
    bool mode_found = false;
    double similarity, time;
    list <tuple<string, double, double>> results;
//    vector <Hash> hashes = generateHashes(t, seed);
    uint64 *hashes_b = generateHashes_b(t, seed);
//    for (int i = 0; i < t; i++) {
//        cout << "hashes_b[i]: " << hashes_b[i] << endl;
//    }

    // GET HASH VALUES LIST
    /*uint64 *list1[t];
    uint64 *list2[t];
    for (int i = 0; i < t; i++) {
        list1[i] = (uint64 *) malloc(sizeof(uint64) * (sequence1.size() - k + 1));
        list2[i] = (uint64 *) malloc(sizeof(uint64) * (sequence2.size() - k + 1));
        for (int j = 0; j < sequence1.size() - k + 1; j++) {
            list1[i][j] = UINT64_MAX;
        }
        for (int j = 0; j < sequence2.size() - k + 1; j++) {
            list2[i][j] = UINT64_MAX;
        }
    }
    getList(k, list1, sequence1, hashes);
    getList(k, list2, sequence2, hashes);*/

    if (cal_name == "all" || cal_name == "minhash_parallel") {
        if (t < 1) {
            cerr << endl;
            cerr << "You must provide a parameter --t=POSITIVE_INTEGER parameter for minhash modes!" << endl << endl;
            exit(1);
        }
        mode_found = true;
        ini_time = clock();
        // vector<Hash> hashes = generateHashes(t, seed);
        vector <vector<uint64>> sig1 = genSig(k, m, t, dnaList1, sequence1.size(), hashes_b);
        vector <vector<uint64>> sig2 = genSig(k, m, t, dnaList2, sequence2.size(), hashes_b);
        cout << "sig1:  size:" << sig1[0].size() << endl;
        output_signature(sig1);
        cout << "\nsig2:  size:" << sig2[0].size() << endl;
        output_signature(sig2);
        cout << endl;
        similarity = computeSim(sig1, sig2);
        time = double(clock() - ini_time) / CLOCKS_PER_SEC;
        results.emplace_back("minhash_parallel", similarity, time);
    }
    if (!mode_found) usage();

    // OUTPUT RESULTS
    if (e) {
        cout << setw(12) << "cal_name" << setw(14) << "seed" << setw(5) << "k" << setw(5) << "m" << setw(7) << "t";
        cout << setw(13) << fixed << "time" << setw(13) << fixed << "similarity" << endl;
    } else {
        cout << "===========================" << endl;
        cout << "k:" << k << setw(7) << fixed << "m:" << m << setw(7) << fixed << "t:" << t << endl;
        cout << "===========================" << endl;
    }
    cout.precision(8);
    for (auto &result : results) {
        if (e) {
            cout << setw(12) << get<0>(result) << setw(14) << seed << setw(5) << k << setw(5) << m << setw(7) << t;
            cout << setw(13) << fixed << get<2>(result) << setw(13) << fixed << get<1>(result) << endl;
        } else {
            cout << uline_on << get<0>(result) << uline_off << endl;
            cout << "time: " << setw(21) << fixed << get<2>(result) << endl;
            cout << "similarity: " << setw(15) << fixed << get<1>(result) << endl;
            cout << "===========================" << endl;
        }
    }

}
